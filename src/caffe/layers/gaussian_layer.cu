#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.
// Gaussian neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void GaussianForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
	  Dtype x = in[index];
	  out[index] = exp(Dtype(-1)*x*x/Dtype(16));
    //Dtype exp2x = exp(2*in[index]);
   // out[index] = (exp2x - Dtype(1))/(exp2x + Dtype(1));
  }
}

template <typename Dtype>
Dtype GaussianLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  GaussianForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
  return Dtype(0);
}

template <typename Dtype>
__global__ void GaussianBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype inx = in_data[index];
	Dtype exp2x = exp(Dtype(-1)*inx*inx/Dtype(16));
	out_diff[index] = in_diff[index] * (exp2x*Dtype(-2)*inx/Dtype(16));
    //Dtype exp2x = exp(2*in_data[index]);
    //Dtype tanhx = (exp2x - Dtype(1))/(exp2x + Dtype(1));
    //out_diff[index] = in_diff[index] * (1 - tanhx*tanhx);
  }
}

template <typename Dtype>
void GaussianLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    GaussianBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_CLASS(GaussianLayer);


}  // namespace caffe
